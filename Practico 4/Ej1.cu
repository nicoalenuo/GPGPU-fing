#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include <iostream>

using namespace std;

#define FILAS 64
#define COLUMNAS 64
#define TAM_BLOQUE_X 32
#define TAM_BLOQUE_Y 32

#define CUDA_CHK(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true) {
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ void transpose_kernel(int* d_matrix, int* d_transposed_matrix) {
	extern __shared__ int shared_block[];

	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	const int x_shared = threadIdx.x;
	const int y_shared = threadIdx.y;

	const int id = COLUMNAS * y + x;
	const int id_transpuesta = (blockIdx.x * blockDim.x * COLUMNAS + blockIdx.y * blockDim.y) + y_shared * COLUMNAS + x_shared;

	shared_block[x_shared * blockDim.x + y_shared] = d_matrix[id];

	__syncthreads();

	d_transposed_matrix[id_transpuesta] = shared_block[y_shared * blockDim.x + x_shared];
}

// Es el mismo codigo que transpose_kernel
// pero para moverse sobre la matrix "shared_block"
// se utiliza blockDim.x + 1 como tamaño de fila 
__global__ void transpose_kernel_dummy_column(int* d_matrix, int* d_transposed_matrix) {
	extern __shared__ int shared_block[];

	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	const int x_shared = threadIdx.x;
	const int y_shared = threadIdx.y;

	const int id = COLUMNAS * y + x;
	const int id_transpuesta = (blockIdx.x * blockDim.x * COLUMNAS + blockIdx.y * blockDim.y) + y_shared * COLUMNAS + x_shared;

	shared_block[x_shared * (blockDim.x + 1) + y_shared] = d_matrix[id];

	__syncthreads();

	d_transposed_matrix[id_transpuesta] = shared_block[y_shared * (blockDim.x + 1) + x_shared];
}

int main(int argc, char* argv[]) {
	size_t tam_matriz = FILAS * COLUMNAS * sizeof(int);

	int* h_matrix;
	int* d_matrix;
	int* d_transposed_matrix;

	h_matrix = (int*)malloc(tam_matriz);

	for (int i = 0; i < FILAS * COLUMNAS; i++)
		h_matrix[i] = i;

	CUDA_CHK(hipMalloc((void**)&d_matrix, tam_matriz));
	CUDA_CHK(hipMalloc((void**)&d_transposed_matrix, tam_matriz));

	CUDA_CHK(hipMemcpy(d_matrix, h_matrix, tam_matriz, hipMemcpyHostToDevice));

	dim3 tamGrid1(FILAS / TAM_BLOQUE_X, COLUMNAS / TAM_BLOQUE_Y);
	dim3 tamBlock1(TAM_BLOQUE_X, TAM_BLOQUE_Y);

	//for (int i = 0; i < 10; i++) {
		
		transpose_kernel << <tamGrid1, tamBlock1, TAM_BLOQUE_X * TAM_BLOQUE_Y * sizeof(int) >> > (d_matrix, d_transposed_matrix);
		CUDA_CHK(hipGetLastError());
		CUDA_CHK(hipDeviceSynchronize());
		
		transpose_kernel_dummy_column << <tamGrid1, tamBlock1, (TAM_BLOQUE_X + 1) * TAM_BLOQUE_Y * sizeof(int) >> > (d_matrix, d_transposed_matrix);
		CUDA_CHK(hipGetLastError());
		CUDA_CHK(hipDeviceSynchronize());
		
	//}

	CUDA_CHK(hipMemcpy(h_matrix, d_transposed_matrix, tam_matriz, hipMemcpyDeviceToHost));

	for (int i = 0; i < FILAS; i++) {
		for (int j = 0; j < COLUMNAS; j++) {
			cout << h_matrix[i * COLUMNAS + j] << " ";
		}
		cout << endl;
	}

	free(h_matrix);
	CUDA_CHK(hipFree(d_matrix));
	CUDA_CHK(hipFree(d_transposed_matrix));

	return 0;
}


