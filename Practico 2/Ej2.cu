#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
  
#define A 15
#define B 27
#define M 256
#define A_MMI_M -17

#define CANT_BLOQUES 5
#define CANT_HILOS 1024

#define N 512

#define CUDA_CHK(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

void read_file(const char*, int*);
int get_text_length(const char* fname);

__device__ int modulo(int a, int b) {
	int r = a % b;
	r = (r < 0) ? r + b : r;
	return r;
}

__global__ void decrypt_kernel(int* d_message, int length) {
	const int id = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = id; i < length; i += gridDim.x * blockDim.x) {
		char letraEncriptada = d_message[i];
		char letraDesencriptada = modulo(A_MMI_M * (letraEncriptada - B), M);
		d_message[i] = letraDesencriptada;
	}
}

__global__ void count_kernel(int* d_message, int* d_counter, int length) {
	const int id = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = id; i < length; i += gridDim.x * blockDim.x) {
		atomicAdd(&d_counter[d_message[i]], 1); 
	}
}



int* count_letters(int* d_message, int length) {
	int* h_counter;
	int* d_counter;

	h_counter = (int*)malloc(256 * sizeof(int));

	CUDA_CHK(hipMalloc((void**)&d_counter, 256 * sizeof(int)));
	CUDA_CHK(hipMemset(d_counter, 0, 256 * sizeof(int)));

	dim3 tamGrid(CANT_BLOQUES, 1);
	dim3 tamBlock(CANT_HILOS, 1, 1);

	count_kernel << <tamGrid, tamBlock >> > (d_message, d_counter, length);
	CUDA_CHK(hipGetLastError());
	CUDA_CHK(hipDeviceSynchronize());

	CUDA_CHK(hipMemcpy(h_counter, d_counter, 256 * sizeof(int), hipMemcpyDeviceToHost));

	CUDA_CHK(hipFree(d_counter));

	return h_counter;
}

int main(int argc, char* argv[]) {
	int* h_message;
	int* d_message;

	unsigned int size;

	const char* fname;

	if (argc < 2) printf("Debe ingresar el nombre del archivo\n");
	else
		fname = argv[1];

	int length = get_text_length(fname);
	size = length * sizeof(int);

	h_message = (int*)malloc(size);
	read_file(fname, h_message);

	CUDA_CHK(hipMalloc((void**)&d_message, size));
	CUDA_CHK(hipMemcpy(d_message, h_message, size, hipMemcpyHostToDevice));

	dim3 tamGrid(CANT_BLOQUES, 1);
	dim3 tamBlock(CANT_HILOS, 1, 1);

	decrypt_kernel <<<tamGrid, tamBlock >>> (d_message, length);
	CUDA_CHK(hipGetLastError());
	CUDA_CHK(hipDeviceSynchronize());


	CUDA_CHK(hipMemcpy(h_message, d_message, size, hipMemcpyDeviceToHost));

	for (int i = 0; i < length; i++) {
		printf("%c", (char)h_message[i]);
	}
	printf("\n");

	int* h_counter = count_letters(d_message, length);

	for (int i = 0; i < 256; i++) {
		printf("%c: %i ", char(i), h_counter[i]);
	}
	printf("\n");

	CUDA_CHK(hipFree(d_message));
	free(h_message);
	free(h_counter);

	return 0;
}


int get_text_length(const char* fname)
{
	FILE* f = NULL;
	f = fopen(fname, "r"); //read and binary flags

	size_t pos = ftell(f);
	fseek(f, 0, SEEK_END);
	size_t length = ftell(f);
	fseek(f, pos, SEEK_SET);

	fclose(f);

	return length;
}

void read_file(const char* fname, int* input)
{
	// printf("leyendo archivo %s\n", fname );

	FILE* f = NULL;
	f = fopen(fname, "r"); //read and binary flags
	if (f == NULL) {
		fprintf(stderr, "Error: Could not find %s file \n", fname);
		exit(1);
	}

	//fread(input, 1, N, f);
	int c;
	while ((c = getc(f)) != EOF) {
		*(input++) = c;
	}

	fclose(f);
}
